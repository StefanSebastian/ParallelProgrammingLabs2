#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

using namespace cv;
using namespace std;


#define THREADS_PER_BLOCK 1024

// serial
Mat rotateImage(Mat image, double theta) {
	float rads = (theta * 3.14159265) / 180.0;

	int r0 = image.rows / 2;
	int c0 = image.cols / 2;

	Mat newImage(image.rows, image.cols, CV_8UC3, Scalar(0, 0, 0));
	for (int r = 0; r < image.rows; r++)
	{
		for (int c = 0; c < image.cols; c++)
		{
			// calculate new coordinates
			int r1 = (int)(r0 + ((r - r0) * cos(rads)) - ((c - c0) * sin(rads)));
			int c1 = (int)(c0 + ((r - r0) * sin(rads)) + ((c - c0) * cos(rads)));

			// set pixel
			if (r1 >= 0 && r1 < image.rows && c1 >= 0 && c1 < image.cols) {
				newImage.at<Vec3b>(r1, c1) = image.at<Vec3b>(r, c);
			}
		}
	}

	return newImage;
}



// parallel
__global__ void rotate(int* pixelPositions, int* rows, int* cols, double* sinOfRads, double* cosOfRads) {
	// get position
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int r = (int)(index / *cols);
	int c = index % *cols;

	int r0 = *rows / 2;
	int c0 = *cols / 2;

	// calculate new coordinates
	int r1 = (int)(r0 + ((r - r0) * *cosOfRads) - ((c - c0) * *sinOfRads));
	int c1 = (int)(c0 + ((r - r0) * *sinOfRads) + ((c - c0) * *cosOfRads));

	// set pixel
	if (r1 >= 0 && r1 < *rows && c1 >= 0 && c1 < *cols) {
		pixelPositions[index] = r1 * (*cols) + c1;
	}

}

Mat parallelOperations(Mat image, double theta) {
	// init values
	float rads = (theta * 3.14159265) / 180.0;
	double sinOfRads = sin(rads);
	double cosOfRads = cos(rads);
	int rows = image.rows;
	int cols = image.cols;
	Mat newImage(image.rows, image.cols, CV_8UC3, Scalar(0, 0, 0));

	int* pixelPositions = (int*)malloc(image.rows * image.cols * sizeof(int));

	int *d_pixelPositions; double *d_sinOfRads, *d_cosOfRads; int *d_rows, *d_cols;  // device copies

	// Alloc space for device copies
	hipError_t err;
	err = hipMalloc((void **)&d_pixelPositions, image.rows * image.cols * sizeof(int));
	if (err != hipSuccess) {
		cout << "alloc error";
	}
	err = hipMalloc((void **)&d_sinOfRads, sizeof(double));
	if (err != hipSuccess) {
		cout << "alloc error";
	}
	err = hipMalloc((void **)&d_cosOfRads, sizeof(double));
	if (err != hipSuccess) {
		cout << "alloc error";
	}
	err = hipMalloc((void **)&d_rows, sizeof(int));
	if (err != hipSuccess) {
		cout << "alloc error";
	}
	err = hipMalloc((void **)&d_cols, sizeof(int));
	if (err != hipSuccess) {
		cout << "alloc error";
	}

	// Copy inputs to device
	err = hipMemcpy(d_pixelPositions, &image, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "alloc error";
	}
	err = hipMemcpy(d_sinOfRads, &sinOfRads, sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "alloc error";
	}
	err = hipMemcpy(d_cosOfRads, &cosOfRads, sizeof(double), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "alloc error";
	}
	err = hipMemcpy(d_rows, &rows, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "alloc error";
	}
	err = hipMemcpy(d_cols, &cols, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "alloc error";
	}
	
	// Launch rotate() kernel on GPU 
	rotate <<<(image.rows * image.cols + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(d_pixelPositions, d_rows, d_cols, d_sinOfRads, d_cosOfRads);
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		cout << "sync error";
	}

	// Copy result back to host
	err = hipMemcpy(pixelPositions, d_pixelPositions, image.rows * image.cols * sizeof(int), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		cout << "alloc error";
	}

	// Cleanup
	hipFree(d_pixelPositions); hipFree(d_sinOfRads); hipFree(d_cosOfRads); hipFree(d_rows); hipFree(d_cols);

	for (int i = 0; i < image.rows * image.cols; i++) {
		int rowFirstMat = i / image.cols;
		int colFirstMat = i % image.cols;

		int rowNewMat = pixelPositions[i] / image.cols;
		int colNewMat = pixelPositions[i] % image.cols;

		//cout << rowFirstMat << " " << colFirstMat << endl;
		//cout << rowNewMat << " " << colNewMat << endl;

		if (rowNewMat >= 0 && rowNewMat < image.rows && colNewMat >= 0 && colNewMat < image.cols) {
			newImage.at<Vec3b>(rowNewMat, colNewMat) = image.at<Vec3b>(rowFirstMat, colFirstMat);
		}
	}

	return newImage;
}


void processImg() {
	Mat image;
	image = imread("mitzi.bmp", CV_LOAD_IMAGE_COLOR);   // Read the file

	if (!image.data)                              // Check for invalid input
	{
		cout << "Could not open or find the image" << std::endl;
		return;
	}

	cout << "Input theta : ";
	double theta;
	cin >> theta;

	Mat rotated = parallelOperations(image, theta);

	namedWindow("Display window", WINDOW_AUTOSIZE);// Create a window for display.
	imshow("Display window", rotated);                   // Show our image inside it.
	waitKey(0);                                          // Wait for a keystroke in the window
}


int main(void) {
	processImg();
}